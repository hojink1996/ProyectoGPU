#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "operations.cuh"
#define BLOCK_SIZE 512

__global__ void addKernel(const float* a, const float* b, float* c)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	c[i] = a[i] + b[i];
}

__global__ void multiplyKernel(const float* firstInput, const float* secondInput, float* output)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	output[index] = firstInput[index] * secondInput[index];
}

__global__ void maskedAddKernel(const float* firstInput, const float* secondInput, const int* mask, float* output)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	output[index] = firstInput[index] + secondInput[index] * mask[index];
}

__global__ void softplusKernel(const float* input, float* output)
{
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	output[index] = logf(1 + expf(input[index]));
}

__global__ void dotProductCuda(const float* firstInput, const float* secondInput, float* output)
{
	// Dynamically allocate the share memory
	int THREADS_PER_BLOCK = __cudaGet_blockDim().x;
	__shared__ float sharedMemory[BLOCK_SIZE];
	int multiplyX = blockDim.x * blockIdx.x + threadIdx.x;
	sharedMemory[threadIdx.x] = firstInput[multiplyX] * secondInput[multiplyX];

	// All of the threads must be done with the multiplication
	__syncthreads();

	// Add the values in the block
	if (threadIdx.x == 0)
	{
		float blockProduct = 0;
		for (int i = 0; i < THREADS_PER_BLOCK; ++i)
		{
			blockProduct += sharedMemory[i];
			// printf("%s", sharedMemory[i]);
		}
		atomicAdd(output, blockProduct);
	}
}

__global__ void dotProductWindowCuda(const float* firstInput, const float* secondInput, const int N, float* output)
{
	// Dynamically allocate the share memory
	//int THREADS_PER_BLOCK = __cudaGet_blockDim().x;
	__shared__ float sharedMemory[BLOCK_SIZE];
	int multiplyX = blockDim.x * blockIdx.x + threadIdx.x;
	sharedMemory[threadIdx.x] = firstInput[multiplyX] * secondInput[multiplyX % N];
	//printf(" (%i) %f * %f  = %f \n", multiplyX, firstInput[multiplyX], secondInput[multiplyX % N], sharedMemory[threadIdx.x]);

	int tIdx = threadIdx.x;

	// All of the threads must be done with the multiplication
	__syncthreads();

	// Add the values in the block
	int offset = blockDim.x * blockIdx.x;
	if (threadIdx.x == 0)
	{
		for (int i = 0; i < BLOCK_SIZE; ++i)
		{
			output[(offset + i) / N] += sharedMemory[i];
		 }
	}	
	__syncthreads();

	//printf("(%i) out: %f, %f, %f, %f\n", multiplyX, output[0], output[1], output[2], output[3]);
}

namespace CudaFunctions
{
	void dotProductWindow(float* firstInput, float* secondInput, int N, float* output, int outputSize)
	{
		int inputSize = N * outputSize;
		// Allocate memory for the CUDA operations
		float* dFirstInput;
		float* dSecondInput;
		float* dOutput;
		hipMalloc((void**)& dFirstInput, inputSize * sizeof(float));
		hipMalloc((void**)& dSecondInput, N * sizeof(float));
		hipMalloc((void**)& dOutput, outputSize * sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dFirstInput, firstInput, inputSize * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dSecondInput, secondInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dOutput, output, outputSize * sizeof(float), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid((inputSize + BLOCK_SIZE - 1)/ BLOCK_SIZE);

		// Run the kernel
		dotProductWindowCuda<<<dimGrid, dimBlock>>> (dFirstInput, dSecondInput, N, dOutput);
		hipDeviceSynchronize();

		// Copy the values to host
		hipMemcpy(output, dOutput, outputSize * sizeof(float), hipMemcpyDeviceToHost);

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR CUDA: %s \n", hipGetErrorString(error));
			return;
		}	

		// Clean up the values
		hipFree(dFirstInput); hipFree(dSecondInput); hipFree(dOutput);
	}

	float dotProduct(float* firstInput, float* secondInput, int N)
	{
		// Allocate memory for the CUDA operations
		float output;
		float* dFirstInput;
		float* dSecondInput;
		float* dOutput;
		hipMalloc((void**)&dFirstInput, N * sizeof(float));
		hipMalloc((void**)&dSecondInput, N * sizeof(float));
		hipMalloc((void**)&dOutput, sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dFirstInput, firstInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dSecondInput, secondInput, N * sizeof(float), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(N / BLOCK_SIZE + 1);

		// Run the kernel
		dotProductCuda << <dimGrid, dimBlock >> > (dFirstInput, dSecondInput, dOutput);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
			return 1;
		}

		// Copy the values to host
		hipMemcpy(&output, dOutput, sizeof(float), hipMemcpyDeviceToHost);

		// Clean up the values
		hipFree(dFirstInput); hipFree(dSecondInput); hipFree(dOutput);

		return output;
	}

	void add(float* firstInput, float* secondInput, int N, float* output)
	{
		// Allocate memory for the CUDA operations
		float* dFirstInput;
		float* dSecondInput;
		float* dOutput;
		hipMalloc((void**)& dFirstInput, N * sizeof(float));
		hipMalloc((void**)& dSecondInput, N * sizeof(float));
		hipMalloc((void**)& dOutput, N * sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dFirstInput, firstInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dSecondInput, secondInput, N * sizeof(float), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(N);
		dim3 dimGrid(N / BLOCK_SIZE + 1);

		// Run the kernel
		addKernel <<<dimGrid, dimBlock >>> (dFirstInput, dSecondInput, dOutput);
		hipDeviceSynchronize();

		// Copy the values to host
		hipMemcpy(output, dOutput, N * sizeof(float), hipMemcpyDeviceToHost);
		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
			return;
		}
		// Clean up the values
		hipFree(dFirstInput); hipFree(dSecondInput); hipFree(dOutput);
	}

	void multiply(float* firstInput, float* secondInput, int N, float* output)
	{
		// Allocate memory for the CUDA operations
		float* dFirstInput;
		float* dSecondInput;
		float* dOutput;
		hipMalloc((void**)& dFirstInput, N * sizeof(float));
		hipMalloc((void**)& dSecondInput, N * sizeof(float));
		hipMalloc((void**)& dOutput, N * sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dFirstInput, firstInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dSecondInput, secondInput, N * sizeof(float), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(N / BLOCK_SIZE + 1);

		// Run the kernel
		multiplyKernel <<<dimGrid, dimBlock >>> (dFirstInput, dSecondInput, dOutput);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
			return;
		}

		// Copy the values to host
		hipMemcpy(output, dOutput, N * sizeof(float), hipMemcpyDeviceToHost);

		// Clean up the values
		hipFree(dFirstInput); hipFree(dSecondInput); hipFree(dOutput);
	}

	void maskedAdd(float* firstInput, float* secondInput, int* mask, int N, float* output)
	{
		// Allocate memory for the CUDA operations
		float* dFirstInput;
		float* dSecondInput;
		float* dOutput;
		int* dMask;
		hipMalloc((void**)& dFirstInput, N * sizeof(float));
		hipMalloc((void**)& dSecondInput, N * sizeof(float));
		hipMalloc((void**)& dMask, N * sizeof(int));
		hipMalloc((void**)& dOutput, N * sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dFirstInput, firstInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dSecondInput, secondInput, N * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(dMask, mask, N * sizeof(int), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(N / BLOCK_SIZE + 1);

		// Run the kernel
		maskedAddKernel<<<dimGrid, dimBlock >>> (dFirstInput, dSecondInput, dMask, dOutput);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
			return;
		}

		// Copy the values to host
		hipMemcpy(output, dOutput, N * sizeof(float), hipMemcpyDeviceToHost);

		// Clean up the values
		hipFree(dFirstInput); hipFree(dSecondInput); hipFree(dOutput);
	}


	void softplus(float* input, int N,  float* output)
	{
		// Allocate memory for the CUDA operations
		float* dInput;
		float* dOutput;
		hipMalloc((void**)& dInput, N * sizeof(float));
		hipMalloc((void**)& dOutput, N * sizeof(float));

		// Copy the values to GPU
		hipMemcpy(dInput, input, N * sizeof(float), hipMemcpyHostToDevice);

		// Set the blocks to use
		dim3 dimBlock(BLOCK_SIZE);
		dim3 dimGrid(N / BLOCK_SIZE + 1);

		// Run the kernel
		softplusKernel<<<dimGrid, dimBlock>>> (dInput, dOutput);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if (error != hipSuccess) {
			fprintf(stderr, "ERROR: %s \n", hipGetErrorString(error));
			return;
		}

		// Copy the values to host
		hipMemcpy(output, dOutput, N * sizeof(float), hipMemcpyDeviceToHost);

		// Clean up the values
		hipFree(dInput);  hipFree(dOutput);
	}
}
